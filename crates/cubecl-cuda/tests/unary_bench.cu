
#include <hip/hip_runtime.h>
#include <mma.h>
typedef unsigned char uint8;
typedef unsigned short uint16;
typedef unsigned int uint;
typedef unsigned long long int uint64;
typedef long long int int64;

struct __align__(16) float_4 {
  float i_0;
  float i_1;
  float i_2;
  float i_3;
};

extern "C" __global__ void execute_unary_kernel_f32(float_4 input_0[],
                                                    float_4 input_1[],
                                                    float_4 output_0[],
                                                    uint info[]) {

  int3 absoluteIdx = make_int3(blockIdx.x * blockDim.x + threadIdx.x,
                               blockIdx.y * blockDim.y + threadIdx.y,
                               blockIdx.z * blockDim.z + threadIdx.z);

  uint idxGlobal =
      (absoluteIdx.z * gridDim.x * blockDim.x * gridDim.y * blockDim.y) +
      (absoluteIdx.y * gridDim.x * blockDim.x) + absoluteIdx.x;
  uint l_0_0;
  bool l_0_1;
  bool l_0_2;
  float_4 l_0_3;
  float_4 l_0_4;
  l_0_0 = info[uint(5)];
  l_0_1 = idxGlobal < l_0_0;
  if (l_0_1) {

    for (uint l_2_0 = uint(0); l_2_0 < uint(256); ++l_2_0) {
      l_0_0 = l_2_0 % uint(2);
      l_0_2 = l_0_0 == uint(0);
      if (l_0_2) {
        uint l_3_0;
        l_3_0 = info[uint(0)];
        l_0_3 = (idxGlobal < l_3_0) ? input_0[idxGlobal] : float_4{};
        uint l_3_1;
        l_3_1 = info[uint(1)];
        l_0_4 = (idxGlobal < l_3_1) ? input_1[idxGlobal] : float_4{};
        l_0_4 = float_4{
            l_0_3.i_0 * l_0_4.i_0,
            l_0_3.i_1 * l_0_4.i_1,
            l_0_3.i_2 * l_0_4.i_2,
            l_0_3.i_3 * l_0_4.i_3,
        };
        l_0_4 = float_4{
            cos(l_0_4.i_0),
            cos(l_0_4.i_1),
            cos(l_0_4.i_2),
            cos(l_0_4.i_3),
        };
        uint l_3_2;
        l_3_2 = info[uint(2)];
        l_0_3 = (idxGlobal < l_3_2) ? output_0[idxGlobal] : float_4{};
        l_0_3 = float_4{
            l_0_3.i_0 - l_0_4.i_0,
            l_0_3.i_1 - l_0_4.i_1,
            l_0_3.i_2 - l_0_4.i_2,
            l_0_3.i_3 - l_0_4.i_3,
        };
        uint l_3_3;
        bool l_3_4;
        l_3_3 = info[uint(2)];
        l_3_4 = idxGlobal < l_3_3;
        if (l_3_4) {
          output_0[idxGlobal] = l_0_3;
        }
      } else {
        uint l_3_0;
        l_3_0 = info[uint(0)];
        l_0_4 = (idxGlobal < l_3_0) ? input_0[idxGlobal] : float_4{};
        uint l_3_1;
        l_3_1 = info[uint(1)];
        l_0_3 = (idxGlobal < l_3_1) ? input_1[idxGlobal] : float_4{};
        l_0_4 = float_4{
            l_0_4.i_0 * l_0_3.i_0,
            l_0_4.i_1 * l_0_3.i_1,
            l_0_4.i_2 * l_0_3.i_2,
            l_0_4.i_3 * l_0_3.i_3,
        };
        l_0_4 = float_4{
            cos(l_0_4.i_0),
            cos(l_0_4.i_1),
            cos(l_0_4.i_2),
            cos(l_0_4.i_3),
        };
        uint l_3_2;
        l_3_2 = info[uint(2)];
        l_0_3 = (idxGlobal < l_3_2) ? output_0[idxGlobal] : float_4{};
        l_0_3 = float_4{
            l_0_3.i_0 + l_0_4.i_0,
            l_0_3.i_1 + l_0_4.i_1,
            l_0_3.i_2 + l_0_4.i_2,
            l_0_3.i_3 + l_0_4.i_3,
        };
        uint l_3_3;
        bool l_3_4;
        l_3_3 = info[uint(2)];
        l_3_4 = idxGlobal < l_3_3;
        if (l_3_4) {
          output_0[idxGlobal] = l_0_3;
        }
      }
    }
  }
}