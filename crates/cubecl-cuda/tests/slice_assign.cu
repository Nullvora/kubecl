
#include <hip/hip_runtime.h>
typedef unsigned char uint8;
typedef unsigned short uint16;
typedef unsigned int uint;
typedef unsigned long long int uint64;
typedef long long int int64;

extern "C" __global__ void slice_assign_kernel(float input_0[],
                                               float output_0[], uint info[]) {

  int threadIdxGlobal = threadIdx.x + threadIdx.y * blockDim.x +
                        threadIdx.z * (blockDim.x * blockDim.y);
  bool l_0_0;
  float l_0_1;
  l_0_0 = threadIdxGlobal == uint(0);
  if (l_0_0) {
    uint l_1_0;
    l_1_0 = info[uint(1)];
    const uint slice_1_0_length = min(l_1_0, uint(3)) - uint(2);
    float *slice_1_0 = output_0 + uint(2);
    uint l_1_1;
    bool l_1_2;
    float l_1_3;
    l_1_1 = info[uint(0)];
    l_1_2 = uint(0) < l_1_1;
    l_1_3 = input_0[uint(0)];
    l_0_1 = (l_1_2) ? l_1_3 : float(0.0);
    uint l_1_4;
    bool l_1_5;
    l_1_4 = slice_1_0_length;
    l_1_5 = uint(0) < l_1_4;
    if (l_1_5) {
      slice_1_0[uint(0)] = l_0_1;
    }
  }
}
