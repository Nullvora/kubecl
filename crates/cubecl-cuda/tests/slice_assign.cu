
#include <hip/hip_runtime.h>
typedef unsigned char uint8;
typedef unsigned short uint16;
typedef unsigned int uint;
typedef unsigned long long int uint64;
typedef long long int int64;

extern "C" __global__ void kernel(float input_0[], float output_0[],
                                  uint info[]) {

  int threadIdxGlobal = threadIdx.x + threadIdx.y * blockDim.x +
                        threadIdx.z * (blockDim.x * blockDim.y);
  bool l_0_0;
  float l_0_1;
  l_0_0 = threadIdxGlobal == uint(0);
  if (l_0_0) {
    const uint slice_1_0_length = uint(3) - uint(2);
    float *slice_1_0 = output_0 + uint(2);
    uint l_1_0;
    l_1_0 = info[uint(0)];
    l_0_1 = (uint(0) < l_1_0) ? input_0[uint(0)] : float(0);
    uint l_1_1;
    bool l_1_2;
    l_1_1 = slice_1_0_length;
    l_1_2 = uint(0) < l_1_1;
    if (l_1_2) {
      slice_1_0[uint(0)] = l_0_1;
    }
  }
}